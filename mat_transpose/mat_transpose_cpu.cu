#include "hip/hip_runtime.h"
#include "cuda_header.h"

__global__ void testTime(float *data) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    data[tid] = tid;
}

int main() {
    int n = 1 << 12;

    int blockSize = 1 << 10;

    dim3 block(blockSize);
    dim3 grid((n + blockSize - 1) / blockSize);

    float *h_data = (float *)malloc(sizeof(float) * n);
    memset(h_data, 0, sizeof(float) * n);
    float *d_data = nullptr;
    CHECK(hipMalloc(&d_data, sizeof(float) * n));
    CHECK(
        hipMemcpy(d_data, h_data, sizeof(float) * n, hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    testTime<<<grid, block>>>(d_data);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // 计算时间差
    float milliseconds = 0.f;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Kernel execution time: " << milliseconds << " ms"
              << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_data);
    return 0;
}